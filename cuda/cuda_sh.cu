#include "hip/hip_runtime.h"
/*This file is part of "A Hybrid Parallel Implementation of the Aho-Corasick and Wu-Manber Algorithms Using NVIDIA CUDA and MPI Evaluated on a Biological Sequence Database".

"A Hybrid Parallel Implementation of the Aho-Corasick and Wu-Manber Algorithms Using NVIDIA CUDA and MPI Evaluated on a Biological Sequence Database" is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

"A Hybrid Parallel Implementation of the Aho-Corasick and Wu-Manber Algorithms Using NVIDIA CUDA and MPI Evaluated on a Biological Sequence Database" is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with "A Hybrid Parallel Implementation of the Aho-Corasick and Wu-Manber Algorithms Using NVIDIA CUDA and MPI Evaluated on a Biological Sequence Database".  If not, see <http://www.gnu.org/licenses/>.*/

#include "hip/hip_runtime.h"

texture<int, hipTextureType2D> tex_state_transition;
texture<unsigned int, hipTextureType1D> tex_state_final;
texture<int, hipTextureType1D> tex_bmBc;

//Optimization 4: Group the write to global memory operations
__global__ void sh_kernel5 ( unsigned char *d_text, unsigned int *d_out, int m, int n, int p_size, int alphabet, int numBlocks, int sharedMemSize ) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int r, s;
	
	int i, j, column, matches = 0;
	
	int charactersPerThread = sharedMemSize / blockDim.x;
	
	int startThread = charactersPerThread * threadIdx.x + m - 1;
	int stopThread = startThread + charactersPerThread;
	
	//Define space in shared memory
	extern __shared__ unsigned char s_array[];
	
	//cast data to uint4
	uint4 *uint4_text = reinterpret_cast < uint4 * > ( d_text );
	uint4 uint4_var;
	
	//recast data to uchar4
	uchar4 c0, c4, c8, c12;
	
	for ( int globalMemIndex = blockIdx.x * sharedMemSize; globalMemIndex < n; globalMemIndex += numBlocks * sharedMemSize ) {
	
		for ( i = globalMemIndex/16 + threadIdx.x, j = 0 + threadIdx.x; j < sharedMemSize / 16 && i < n / 16; i+=blockDim.x, j+=blockDim.x ) {
			
			uint4_var = uint4_text[i];
			
			//recast data back to char after the memory transaction
			c0 = *reinterpret_cast<uchar4 *> ( &uint4_var.x );
			c4 = *reinterpret_cast<uchar4 *> ( &uint4_var.y );
			c8 = *reinterpret_cast<uchar4 *> ( &uint4_var.z );
			c12 = *reinterpret_cast<uchar4 *> ( &uint4_var.w );

			s_array[j * 16 + 0] = c0.x;
                        s_array[j * 16 + 1] = c0.y;
                        s_array[j * 16 + 2] = c0.z;
                        s_array[j * 16 + 3] = c0.w;
                        
                        s_array[j * 16 + 4] = c4.x;
                        s_array[j * 16 + 5] = c4.y;
                        s_array[j * 16 + 6] = c4.z;
                        s_array[j * 16 + 7] = c4.w;
                        
                        s_array[j * 16 + 8] = c8.x;
                        s_array[j * 16 + 9] = c8.y;
                        s_array[j * 16 + 10] = c8.z;
                        s_array[j * 16 + 11] = c8.w;
                        
                        s_array[j * 16 + 12] = c12.x;
                        s_array[j * 16 + 13] = c12.y;
                        s_array[j * 16 + 14] = c12.z;
                        s_array[j * 16 + 15] = c12.w;
		}

		//Add m - 1 redundant characters at the end of the shared memory
		//FIXME: optimize this!!
		if ( threadIdx.x < m - 1 )
			s_array[sharedMemSize + threadIdx.x] = d_text[globalMemIndex + sharedMemSize + threadIdx.x];
			
		__syncthreads();
		
		column = startThread;
	
		while ( column < stopThread && globalMemIndex + column < n  ) {
	
			r = 0;
			j = 0;

			while ( j < m && ( s = tex2D ( tex_state_transition, s_array[column - j], r ) ) != -1 ) {
				
				r = s;
				j++;
			}
		
			matches += tex1Dfetch ( tex_state_final, r );
		
			column += tex1Dfetch ( tex_bmBc, d_text[column] );
		}
		
		__syncthreads();
	}
	
	d_out[idx] = matches;
}

extern "C" void cuda_sh5 ( int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_final, int *bmBc ) {

	//Pointer for device memory
	int *d_state_transition, *d_bmBc;
	unsigned int *d_state_final, *d_out;
	
	unsigned char *d_text;

	size_t pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256, sharedMemSize = 16128;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m ) {
		printf("The text size is too small\n");
		exit(1);
	}
	
	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	
	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_bmBc, alphabet * sizeof ( int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_final, state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_bmBc, bmBc, alphabet * sizeof ( int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	
	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_transition, d_state_transition, desc, alphabet, m * p_size + 1, pitch ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_final, d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_bmBc, d_bmBc, alphabet * sizeof ( int ) ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;

	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//Executing kernel in the device
	sh_kernel5<<<dimGrid, dimBlock, sharedMemSize + 16 * ( ( m - 1 ) / 16 + 1 )>>>( d_text, d_out, m, n, p_size, alphabet, numBlocks, sharedMemSize );
	checkCUDAError("kernel invocation");
	
	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	hipEventDestroy ( start );
	hipEventDestroy ( stop );

	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	   
  	//Look at the results
  	int i, matches = 0;
  	
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf ("Kernel 5 matches \t%i\t time \t%f\n", matches, time/1000);
	
	//Free host and device memory
	free ( h_out );

	hipFree ( d_text );
	hipFree ( d_state_transition );
	hipFree ( d_state_final );
	hipFree ( d_bmBc );	
	hipFree ( d_out );
}

//Optimization 3: Read 16 byte words per thread with coalescing. Uint4 words are extracted to shared memory after fetching from global memory so it is not as efficient as extracting on a per thread basis as with the ac uint4 optimization.
__global__ void sh_kernel4 ( unsigned char *d_text, unsigned int *d_out, int m, int n, int p_size, int alphabet, int numBlocks, int sharedMemSize ) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int r, s;
	
	int i, j, column;
	
	int charactersPerThread = sharedMemSize / blockDim.x;
	
	int startThread = charactersPerThread * threadIdx.x + m - 1;
	int stopThread = startThread + charactersPerThread;
	
	//Define space in shared memory
	extern __shared__ unsigned char s_array[];
	
	//cast data to uint4
	uint4 *uint4_text = reinterpret_cast < uint4 * > ( d_text );
	uint4 uint4_var;
	
	//recast data to uchar4
	uchar4 c0, c4, c8, c12;
	
	for ( int globalMemIndex = blockIdx.x * sharedMemSize; globalMemIndex < n; globalMemIndex += numBlocks * sharedMemSize ) {
	
		for ( i = globalMemIndex/16 + threadIdx.x, j = 0 + threadIdx.x; j < sharedMemSize / 16 && i < n / 16; i+=blockDim.x, j+=blockDim.x ) {
			
			uint4_var = uint4_text[i];
			
			//recast data back to char after the memory transaction
			c0 = *reinterpret_cast<uchar4 *> ( &uint4_var.x );
			c4 = *reinterpret_cast<uchar4 *> ( &uint4_var.y );
			c8 = *reinterpret_cast<uchar4 *> ( &uint4_var.z );
			c12 = *reinterpret_cast<uchar4 *> ( &uint4_var.w );

			s_array[j * 16 + 0] = c0.x;
                        s_array[j * 16 + 1] = c0.y;
                        s_array[j * 16 + 2] = c0.z;
                        s_array[j * 16 + 3] = c0.w;
                        
                        s_array[j * 16 + 4] = c4.x;
                        s_array[j * 16 + 5] = c4.y;
                        s_array[j * 16 + 6] = c4.z;
                        s_array[j * 16 + 7] = c4.w;
                        
                        s_array[j * 16 + 8] = c8.x;
                        s_array[j * 16 + 9] = c8.y;
                        s_array[j * 16 + 10] = c8.z;
                        s_array[j * 16 + 11] = c8.w;
                        
                        s_array[j * 16 + 12] = c12.x;
                        s_array[j * 16 + 13] = c12.y;
                        s_array[j * 16 + 14] = c12.z;
                        s_array[j * 16 + 15] = c12.w;
		}

		//Add m - 1 redundant characters at the end of the shared memory
		//FIXME: optimize this!!
		if ( threadIdx.x < m - 1 )
			s_array[sharedMemSize + threadIdx.x] = d_text[globalMemIndex + sharedMemSize + threadIdx.x];
			
		__syncthreads();
		
		column = startThread;
	
		while ( column < stopThread && globalMemIndex + column < n  ) {
	
			r = 0;
			j = 0;

			while ( j < m && ( s = tex2D ( tex_state_transition, s_array[column - j], r ) ) != -1 ) {
				
				r = s;
				j++;
			}
		
			d_out[idx] += tex1Dfetch ( tex_state_final, r );
		
			column += tex1Dfetch ( tex_bmBc, d_text[column] );
		}
		
		__syncthreads();
	}
}

extern "C" void cuda_sh4 ( int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_final, int *bmBc ) {

	//Pointer for device memory
	int *d_state_transition, *d_bmBc;
	unsigned int *d_state_final, *d_out;
	
	unsigned char *d_text;

	size_t pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256, sharedMemSize = 16128;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m ) {
		printf("The text size is too small\n");
		exit(1);
	}
	
	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	
	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_bmBc, alphabet * sizeof ( int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_final, state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_bmBc, bmBc, alphabet * sizeof ( int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	
	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_transition, d_state_transition, desc, alphabet, m * p_size + 1, pitch ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_final, d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_bmBc, d_bmBc, alphabet * sizeof ( int ) ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;

	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//Executing kernel in the device
	sh_kernel4<<<dimGrid, dimBlock, sharedMemSize + 16 * ( ( m - 1 ) / 16 + 1 )>>>( d_text, d_out, m, n, p_size, alphabet, numBlocks, sharedMemSize );
	checkCUDAError("kernel invocation");
	
	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	hipEventDestroy ( start );
	hipEventDestroy ( stop );

	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	   
  	//Look at the results
  	int i, matches = 0;
  	
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf ("Kernel 4 matches \t%i\t time \t%f\n", matches, time/1000);
	
	//Free host and device memory
	free ( h_out );

	hipFree ( d_text );
	hipFree ( d_state_transition );
	hipFree ( d_state_final );
	hipFree ( d_bmBc );	
	hipFree ( d_out );
}

//Optimization 2: Read sharedMemSize characters byte-to-byte from global memory to shared memory to coalescelce memory transactions 
__global__ void sh_kernel3 ( unsigned char *d_text, unsigned int *d_out, int m, int n, int p_size, int alphabet, int numBlocks, int sharedMemSize ) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int r, s;
	
	int i, j, column;
	
	int charactersPerThread = sharedMemSize / blockDim.x;
	
	int startThread = charactersPerThread * threadIdx.x + m - 1;
	int stopThread = startThread + charactersPerThread;
	
	//Define space in shared memory
	extern __shared__ unsigned char s_array[];
	
	for ( int globalMemIndex = blockIdx.x * sharedMemSize; globalMemIndex < n; globalMemIndex += numBlocks * sharedMemSize ) {
	
	
		for ( i = globalMemIndex + threadIdx.x, j = 0 + threadIdx.x; ( j < sharedMemSize + m - 1 && i < n ); i+=blockDim.x, j+=blockDim.x )
			s_array[j] = d_text[i];
			
		__syncthreads();
		
		column = startThread;
	
		while ( column < stopThread && globalMemIndex + column < n  ) {
	
			r = 0;
			j = 0;

			while ( j < m && ( s = tex2D ( tex_state_transition, s_array[column - j], r ) ) != -1 ) {
				
				r = s;
				j++;
			}
		
			d_out[idx] += tex1Dfetch ( tex_state_final, r );
		
			column += tex1Dfetch ( tex_bmBc, d_text[column] );
		}
		
		__syncthreads();
	}
}

extern "C" void cuda_sh3 ( int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_final, int *bmBc ) {

	//Pointer for device memory
	int *d_state_transition, *d_bmBc;
	unsigned int *d_state_final, *d_out;
	
	unsigned char *d_text;

	size_t pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256, sharedMemSize = 16128;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m ) {
		printf("The text size is too small\n");
		exit(1);
	}
	
	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	
	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_bmBc, alphabet * sizeof ( int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_final, state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_bmBc, bmBc, alphabet * sizeof ( int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	
	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_transition, d_state_transition, desc, alphabet, m * p_size + 1, pitch ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_final, d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_bmBc, d_bmBc, alphabet * sizeof ( int ) ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;

	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//Executing kernel in the device
	sh_kernel3<<<dimGrid, dimBlock, sharedMemSize + m - 1>>>( d_text, d_out, m, n, p_size, alphabet, numBlocks, sharedMemSize );
	checkCUDAError("kernel invocation");
	
	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	hipEventDestroy ( start );
	hipEventDestroy ( stop );

	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	   
  	//Look at the results
  	int i, matches = 0;
  	
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf ("Kernel 3 matches \t%i\t time \t%f\n", matches, time/1000);
	
	//Free host and device memory
	free ( h_out );

	hipFree ( d_text );
	hipFree ( d_state_transition );
	hipFree ( d_state_final );
	hipFree ( d_bmBc );
	hipFree ( d_out );
}

//Optimization 1: Use the texture cache for the pattern
__global__ void sh_kernel2 ( unsigned char *d_text, unsigned int *d_out, int m, int n, int p_size, int alphabet, int numBlocks ) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	int charactersPerBlock = n / numBlocks;
	
	int startBlock = blockIdx.x * charactersPerBlock;
	int stopBlock = startBlock + charactersPerBlock;
	
	int charactersPerThread = ( stopBlock - startBlock ) / blockDim.x;
	
	int startThread = startBlock + charactersPerThread * threadIdx.x + m - 1;
	int stopThread = startThread + charactersPerThread;

	int r, s;
	
	int column = startThread, j;
	
	while ( column < stopThread ) {
	
		r = 0;
		j = 0;

		while ( j < m && ( s = tex2D ( tex_state_transition, d_text[column - j], r ) ) != -1 ) {
				
			r = s;
			j++;
		}
		
		d_out[idx] += tex1Dfetch ( tex_state_final, r );
		
		column += tex1Dfetch ( tex_bmBc, d_text[column] );
	}
}

extern "C" void cuda_sh2 ( int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_final, int *bmBc ) {

	//Pointer for device memory
	int *d_state_transition, *d_bmBc;
	unsigned int *d_state_final, *d_out;
	
	unsigned char *d_text;
	
	size_t pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m ) {
		printf("The text size is too small\n");
		exit(1);
	}
	
	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );

	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_bmBc, alphabet * sizeof ( int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_final, state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_bmBc, bmBc, alphabet * sizeof ( int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );

	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_transition, d_state_transition, desc, alphabet, m * p_size + 1, pitch ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_final, d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_bmBc, d_bmBc, alphabet * sizeof ( int ) ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;

	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//Executing kernel in the device
	sh_kernel2<<<dimGrid, dimBlock>>>( d_text, d_out, m, n, p_size, alphabet, numBlocks );
	
	checkCUDAError("kernel invocation");
	
	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	hipEventDestroy ( start );
	hipEventDestroy ( stop );

	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
   
  	//Look at the results
  	int i, matches = 0;
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf ("Kernel 2 matches \t%i\t time \t%f\n", matches, time/1000);
		
	//Free host and device memory
	free ( h_out );
	
	hipFree ( d_text );
	hipFree ( d_state_transition );
	hipFree ( d_state_final );
	hipFree ( d_bmBc );
	hipFree ( d_out );
}

__global__ void sh_kernel1 ( int *d_state_transition, unsigned int *d_state_final, int *d_bmBc, unsigned char *d_text, unsigned int *d_out, size_t pitch, int m, int n, int p_size, int alphabet, int numBlocks ) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int effective_pitch = pitch / sizeof ( int );
	
	int charactersPerBlock = n / numBlocks;
	
	int startBlock = blockIdx.x * charactersPerBlock;
	int stopBlock = startBlock + charactersPerBlock;
	
	int charactersPerThread = ( stopBlock - startBlock ) / blockDim.x;
	
	int startThread = startBlock + charactersPerThread * threadIdx.x + m - 1;
	int stopThread = startThread + charactersPerThread;

	int r, s;
	
	int column = startThread, j;
	
	while ( column < stopThread ) {
	
		r = 0;
		j = 0;

		while ( j < m && ( s = d_state_transition[r * effective_pitch + d_text[column - j]] ) != -1 ) {
				
			r = s;
			j++;
		}
		
		d_out[idx] += d_state_final[r];
		
		column += d_bmBc[d_text[column]];
	}
}
	
extern "C" void cuda_sh1 ( int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_final, int *bmBc ) {

	//Pointer for device memory
	int *d_state_transition, *d_bmBc;
	unsigned int *d_state_final, *d_out;
	
	unsigned char *d_text;

	size_t pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m ) {
		printf("The text size is too small\n");
		exit(1);
	}
	
	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );

	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_bmBc, alphabet * sizeof ( int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_final, state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_bmBc, bmBc, alphabet * sizeof ( int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );

	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;
	
	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//cudaPrintfInit();
	
	//Executing kernel in the device
	sh_kernel1<<<dimGrid, dimBlock>>>( d_state_transition, d_state_final, d_bmBc, d_text, d_out, pitch, m, n, p_size, alphabet, numBlocks );
	
	checkCUDAError("kernel invocation");
	
	//cudaPrintfDisplay(stdout, true);
	//cudaPrintfEnd();

	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	hipEventDestroy ( start );
	hipEventDestroy ( stop );
	
	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	
  	//Look at the results
  	int i, matches = 0;
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf ("Kernel 1 matches \t%i\t time \t%f\n", matches, time/1000);
		
	//Free host and device memory
	free ( h_out );
	
	hipFree ( d_text );
	hipFree ( d_state_transition );
	hipFree ( d_state_final );
	hipFree ( d_bmBc );
	hipFree ( d_out );
}
