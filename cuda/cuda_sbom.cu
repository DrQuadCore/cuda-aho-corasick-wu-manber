#include "hip/hip_runtime.h"
/*This file is part of "A Hybrid Parallel Implementation of the Aho-Corasick and Wu-Manber Algorithms Using NVIDIA CUDA and MPI Evaluated on a Biological Sequence Database".

"A Hybrid Parallel Implementation of the Aho-Corasick and Wu-Manber Algorithms Using NVIDIA CUDA and MPI Evaluated on a Biological Sequence Database" is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

"A Hybrid Parallel Implementation of the Aho-Corasick and Wu-Manber Algorithms Using NVIDIA CUDA and MPI Evaluated on a Biological Sequence Database" is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with "A Hybrid Parallel Implementation of the Aho-Corasick and Wu-Manber Algorithms Using NVIDIA CUDA and MPI Evaluated on a Biological Sequence Database".  If not, see <http://www.gnu.org/licenses/>.*/

#include "hip/hip_runtime.h"

texture<int, hipTextureType2D> tex_state_transition;
texture<int, hipTextureType2D> tex_state_final_multi;
texture<unsigned char, hipTextureType2D> tex_pattern;

__global__ void sbom_kernel5 ( unsigned int *d_state_final_multi, unsigned char *d_text, unsigned int *d_out, size_t final_multi_pitch, int m, int n, int p_size, int alphabet, int numBlocks, int sharedMemSize ) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int effective_final_multi_pitch = final_multi_pitch / sizeof ( int );
	
	int charactersPerThread = sharedMemSize / blockDim.x;
	
	int startThread = charactersPerThread * threadIdx.x + m - 1;
	int stopThread = startThread + charactersPerThread;
	
	//Define space in shared memory
	extern __shared__ unsigned char s_array[];
	
	//cast data to uint4
	uint4 *uint4_text = reinterpret_cast < uint4 * > ( d_text );
	uint4 uint4_var;
	
	//recast data to uchar4
	uchar4 c0, c4, c8, c12;

	int r, s;
	
	int column, i, j, l, matches = 0;
	
	for ( int globalMemIndex = blockIdx.x * sharedMemSize; globalMemIndex < n; globalMemIndex += numBlocks * sharedMemSize ) {
	
		for ( i = globalMemIndex/16 + threadIdx.x, j = 0 + threadIdx.x; j < sharedMemSize / 16 && i < n / 16; i+=blockDim.x, j+=blockDim.x ) {
			
			uint4_var = uint4_text[i];
			
			//recast data back to char after the memory transaction
			c0 = *reinterpret_cast<uchar4 *> ( &uint4_var.x );
			c4 = *reinterpret_cast<uchar4 *> ( &uint4_var.y );
			c8 = *reinterpret_cast<uchar4 *> ( &uint4_var.z );
			c12 = *reinterpret_cast<uchar4 *> ( &uint4_var.w );

			s_array[j * 16 + 0] = c0.x;
                        s_array[j * 16 + 1] = c0.y;
                        s_array[j * 16 + 2] = c0.z;
                        s_array[j * 16 + 3] = c0.w;
                        
                        s_array[j * 16 + 4] = c4.x;
                        s_array[j * 16 + 5] = c4.y;
                        s_array[j * 16 + 6] = c4.z;
                        s_array[j * 16 + 7] = c4.w;
                        
                        s_array[j * 16 + 8] = c8.x;
                        s_array[j * 16 + 9] = c8.y;
                        s_array[j * 16 + 10] = c8.z;
                        s_array[j * 16 + 11] = c8.w;
                        
                        s_array[j * 16 + 12] = c12.x;
                        s_array[j * 16 + 13] = c12.y;
                        s_array[j * 16 + 14] = c12.z;
                        s_array[j * 16 + 15] = c12.w;
		}

		//Add m - 1 redundant characters at the end of the shared memory
		//FIXME: optimize this!!
		if ( threadIdx.x < m - 1 )
			s_array[sharedMemSize + threadIdx.x] = d_text[globalMemIndex + sharedMemSize + threadIdx.x];
			
		__syncthreads();
		
		column = startThread;
	
		while ( column < stopThread ) {
	
			r = 0;
			j = 0;

			while ( j < m && ( s = tex2D ( tex_state_transition, s_array[column - j], r ) ) != -1 ) {
				
				r = s;
				j++;
			}

			if ( j == m && tex2D ( tex_state_final_multi, 0, r ) > 0 ) {

				for ( i = 1; i <= tex2D ( tex_state_final_multi, 0, r ); i++ ) {

					//memcmp implementation
					for ( l = 0; l < m; l++ )
						if ( tex2D ( tex_pattern, l, d_state_final_multi[r * effective_final_multi_pitch + i] ) != s_array[column - m + 1 + l] )
							break;

					if ( l == m ) {
						matches++;
						break;
					}

				}
					column++;
			}
			else	
				column += MAX ( m - j, 1);
		}
		__syncthreads();
	}
	
	d_out[idx] = matches;
}

extern "C" void cuda_sbom5 ( unsigned char *pattern, int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_final_multi ) {

	//Pointer for device memory
	int *d_state_transition;
	unsigned int *d_state_final_multi, *d_out;
	
	unsigned char *d_pattern, *d_text;

	size_t pattern_pitch, transition_pitch, final_multi_pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256, sharedMemSize = 16128;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m ) {
		printf("The text size is too small\n");
		exit(1);
	}

	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );

	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_pattern, &pattern_pitch, m * sizeof ( unsigned char ), p_size ) );
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &transition_pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	checkCudaErrors ( hipMallocPitch ( &d_state_final_multi, &final_multi_pitch, 200 * sizeof ( unsigned int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );

	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_pattern, pattern_pitch, pattern, m * sizeof ( unsigned char ), m * sizeof ( unsigned char ), p_size, hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, transition_pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), m * p_size + 1, hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy2D ( d_state_final_multi, final_multi_pitch, state_final_multi, 200 * sizeof ( int ), 200 * sizeof ( int ), m * p_size + 1, hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc int_desc = hipCreateChannelDesc<int>();
	hipChannelFormatDesc char_desc = hipCreateChannelDesc<unsigned char>();
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_transition, d_state_transition, int_desc, alphabet, m * p_size + 1, transition_pitch ) );
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_final_multi, d_state_final_multi, int_desc, 200, m * p_size + 1, final_multi_pitch ) );
	checkCudaErrors ( hipBindTexture2D ( 0, tex_pattern, d_pattern, char_desc, m, p_size, pattern_pitch ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;
	
	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//cudaPrintfInit();
	
	//Executing kernel in the device
	sbom_kernel5<<<dimGrid, dimBlock, sharedMemSize + 16 * ( ( m - 1 ) / 16 + 1 )>>>( d_state_final_multi, d_text, d_out, final_multi_pitch, m, n, p_size, alphabet, numBlocks, sharedMemSize );
	
	checkCUDAError("kernel invocation");
	
	//cudaPrintfDisplay(stdout, true);
	//cudaPrintfEnd();

	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	hipEventDestroy ( start );
	hipEventDestroy ( stop );
	
	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	
  	//Look at the results
  	int i, matches = 0;
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf ("Kernel 5 matches \t%i\t time \t%f\n", matches, time/1000);
		
	//Free host and device memory
	free ( h_out );
	
	hipFree ( d_text );
	hipFree ( d_pattern );
	hipFree ( d_state_transition );
	hipFree ( d_state_final_multi );
	hipFree ( d_out );
}

__global__ void sbom_kernel4 ( unsigned int *d_state_final_multi, unsigned char *d_text, unsigned int *d_out, size_t final_multi_pitch, int m, int n, int p_size, int alphabet, int numBlocks, int sharedMemSize ) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int effective_final_multi_pitch = final_multi_pitch / sizeof ( int );
	
	int charactersPerThread = sharedMemSize / blockDim.x;
	
	int startThread = charactersPerThread * threadIdx.x + m - 1;
	int stopThread = startThread + charactersPerThread;
	
	//Define space in shared memory
	extern __shared__ unsigned char s_array[];
	
	//cast data to uint4
	uint4 *uint4_text = reinterpret_cast < uint4 * > ( d_text );
	uint4 uint4_var;
	
	//recast data to uchar4
	uchar4 c0, c4, c8, c12;

	int r, s;
	
	int column, i, j, l;
	
	for ( int globalMemIndex = blockIdx.x * sharedMemSize; globalMemIndex < n; globalMemIndex += numBlocks * sharedMemSize ) {
	
		for ( i = globalMemIndex/16 + threadIdx.x, j = 0 + threadIdx.x; j < sharedMemSize / 16 && i < n / 16; i+=blockDim.x, j+=blockDim.x ) {
			
			uint4_var = uint4_text[i];
			
			//recast data back to char after the memory transaction
			c0 = *reinterpret_cast<uchar4 *> ( &uint4_var.x );
			c4 = *reinterpret_cast<uchar4 *> ( &uint4_var.y );
			c8 = *reinterpret_cast<uchar4 *> ( &uint4_var.z );
			c12 = *reinterpret_cast<uchar4 *> ( &uint4_var.w );

			s_array[j * 16 + 0] = c0.x;
                        s_array[j * 16 + 1] = c0.y;
                        s_array[j * 16 + 2] = c0.z;
                        s_array[j * 16 + 3] = c0.w;
                        
                        s_array[j * 16 + 4] = c4.x;
                        s_array[j * 16 + 5] = c4.y;
                        s_array[j * 16 + 6] = c4.z;
                        s_array[j * 16 + 7] = c4.w;
                        
                        s_array[j * 16 + 8] = c8.x;
                        s_array[j * 16 + 9] = c8.y;
                        s_array[j * 16 + 10] = c8.z;
                        s_array[j * 16 + 11] = c8.w;
                        
                        s_array[j * 16 + 12] = c12.x;
                        s_array[j * 16 + 13] = c12.y;
                        s_array[j * 16 + 14] = c12.z;
                        s_array[j * 16 + 15] = c12.w;
		}

		//Add m - 1 redundant characters at the end of the shared memory
		//FIXME: optimize this!!
		if ( threadIdx.x < m - 1 )
			s_array[sharedMemSize + threadIdx.x] = d_text[globalMemIndex + sharedMemSize + threadIdx.x];
			
		__syncthreads();
		
		column = startThread;
	
		while ( column < stopThread ) {
	
			r = 0;
			j = 0;

			while ( j < m && ( s = tex2D ( tex_state_transition, s_array[column - j], r ) ) != -1 ) {
				
				r = s;
				j++;
			}

			if ( j == m && tex2D ( tex_state_final_multi, 0, r ) > 0 ) {

				for ( i = 1; i <= tex2D ( tex_state_final_multi, 0, r ); i++ ) {

					//memcmp implementation
					for ( l = 0; l < m; l++ )
						if ( tex2D ( tex_pattern, l, d_state_final_multi[r * effective_final_multi_pitch + i] ) != s_array[column - m + 1 + l] )
							break;

					if ( l == m ) {
						d_out[idx]++;
						break;
					}

				}
					column++;
			}
			else	
				column += MAX ( m - j, 1);
		}
		__syncthreads();
	}
}

extern "C" void cuda_sbom4 ( unsigned char *pattern, int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_final_multi ) {

	//Pointer for device memory
	int *d_state_transition;
	unsigned int *d_state_final_multi, *d_out;
	
	unsigned char *d_pattern, *d_text;

	size_t pattern_pitch, transition_pitch, final_multi_pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256, sharedMemSize = 16128;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m ) {
		printf("The text size is too small\n");
		exit(1);
	}

	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );

	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_pattern, &pattern_pitch, m * sizeof ( unsigned char ), p_size ) );
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &transition_pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	checkCudaErrors ( hipMallocPitch ( &d_state_final_multi, &final_multi_pitch, 200 * sizeof ( unsigned int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );

	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_pattern, pattern_pitch, pattern, m * sizeof ( unsigned char ), m * sizeof ( unsigned char ), p_size, hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, transition_pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), m * p_size + 1, hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy2D ( d_state_final_multi, final_multi_pitch, state_final_multi, 200 * sizeof ( int ), 200 * sizeof ( int ), m * p_size + 1, hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc int_desc = hipCreateChannelDesc<int>();
	hipChannelFormatDesc char_desc = hipCreateChannelDesc<unsigned char>();
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_transition, d_state_transition, int_desc, alphabet, m * p_size + 1, transition_pitch ) );
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_final_multi, d_state_final_multi, int_desc, 200, m * p_size + 1, final_multi_pitch ) );
	checkCudaErrors ( hipBindTexture2D ( 0, tex_pattern, d_pattern, char_desc, m, p_size, pattern_pitch ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;
	
	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//cudaPrintfInit();
	
	//Executing kernel in the device
	sbom_kernel4<<<dimGrid, dimBlock, sharedMemSize + 16 * ( ( m - 1 ) / 16 + 1 )>>>( d_state_final_multi, d_text, d_out, final_multi_pitch, m, n, p_size, alphabet, numBlocks, sharedMemSize );
	
	checkCUDAError("kernel invocation");
	
	//cudaPrintfDisplay(stdout, true);
	//cudaPrintfEnd();

	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	hipEventDestroy ( start );
	hipEventDestroy ( stop );
	
	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	
  	//Look at the results
  	int i, matches = 0;
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf ("Kernel 4 matches \t%i\t time \t%f\n", matches, time/1000);
		
	//Free host and device memory
	free ( h_out );
	
	hipFree ( d_text );
	hipFree ( d_pattern );
	hipFree ( d_state_transition );
	hipFree ( d_state_final_multi );
	hipFree ( d_out );
}

__global__ void sbom_kernel3 ( unsigned int *d_state_final_multi, unsigned char *d_text, unsigned int *d_out, size_t final_multi_pitch, int m, int n, int p_size, int alphabet, int numBlocks, int sharedMemSize ) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int effective_final_multi_pitch = final_multi_pitch / sizeof ( int );
	
	int charactersPerThread = sharedMemSize / blockDim.x;
	
	int startThread = charactersPerThread * threadIdx.x + m - 1;
	int stopThread = startThread + charactersPerThread;
	
	//Define space in shared memory
	extern __shared__ unsigned char s_array[];

	int r, s;
	
	int column, i, j, l;
	
	for ( int globalMemIndex = blockIdx.x * sharedMemSize; globalMemIndex < n; globalMemIndex += numBlocks * sharedMemSize ) {
		
		for ( i = globalMemIndex + threadIdx.x, j = 0 + threadIdx.x; ( j < sharedMemSize + m - 1 && i < n ); i+=blockDim.x, j+=blockDim.x )
			s_array[j] = d_text[i];
			
		__syncthreads();
		
		column = startThread;
	
		while ( column < stopThread ) {
	
			r = 0;
			j = 0;

			while ( j < m && ( s = tex2D ( tex_state_transition, s_array[column - j], r ) ) != -1 ) {
				
				r = s;
				j++;
			}

			if ( j == m && tex2D ( tex_state_final_multi, 0, r ) > 0 ) {

				for ( i = 1; i <= tex2D ( tex_state_final_multi, 0, r ); i++ ) {

					//memcmp implementation
					for ( l = 0; l < m; l++ )
						if ( tex2D ( tex_pattern, l, d_state_final_multi[r * effective_final_multi_pitch + i] ) != s_array[column - m + 1 + l] )
							break;

					if ( l == m ) {
						d_out[idx]++;
						break;
					}

				}
					column++;
			}
			else	
				column += MAX ( m - j, 1);
		}
		__syncthreads();
	}
}

extern "C" void cuda_sbom3 ( unsigned char *pattern, int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_final_multi ) {

	//Pointer for device memory
	int *d_state_transition;
	unsigned int *d_state_final_multi, *d_out;
	
	unsigned char *d_pattern, *d_text;

	size_t pattern_pitch, transition_pitch, final_multi_pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256, sharedMemSize = 16128;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m ) {
		printf("The text size is too small\n");
		exit(1);
	}

	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );

	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_pattern, &pattern_pitch, m * sizeof ( unsigned char ), p_size ) );
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &transition_pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	checkCudaErrors ( hipMallocPitch ( &d_state_final_multi, &final_multi_pitch, 200 * sizeof ( unsigned int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );

	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_pattern, pattern_pitch, pattern, m * sizeof ( unsigned char ), m * sizeof ( unsigned char ), p_size, hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, transition_pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), m * p_size + 1, hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy2D ( d_state_final_multi, final_multi_pitch, state_final_multi, 200 * sizeof ( int ), 200 * sizeof ( int ), m * p_size + 1, hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc int_desc = hipCreateChannelDesc<int>();
	hipChannelFormatDesc char_desc = hipCreateChannelDesc<unsigned char>();
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_transition, d_state_transition, int_desc, alphabet, m * p_size + 1, transition_pitch ) );
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_final_multi, d_state_final_multi, int_desc, 200, m * p_size + 1, final_multi_pitch ) );
	checkCudaErrors ( hipBindTexture2D ( 0, tex_pattern, d_pattern, char_desc, m, p_size, pattern_pitch ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;
	
	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//cudaPrintfInit();
	
	//Executing kernel in the device
	sbom_kernel3<<<dimGrid, dimBlock, sharedMemSize + m - 1>>>( d_state_final_multi, d_text, d_out, final_multi_pitch, m, n, p_size, alphabet, numBlocks, sharedMemSize );
	
	checkCUDAError("kernel invocation");
	
	//cudaPrintfDisplay(stdout, true);
	//cudaPrintfEnd();

	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	hipEventDestroy ( start );
	hipEventDestroy ( stop );
	
	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	
  	//Look at the results
  	int i, matches = 0;
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf ("Kernel 3 matches \t%i\t time \t%f\n", matches, time/1000);
		
	//Free host and device memory
	free ( h_out );
	
	hipFree ( d_text );
	hipFree ( d_pattern );
	hipFree ( d_state_transition );
	hipFree ( d_state_final_multi );
	hipFree ( d_out );
}

__global__ void sbom_kernel2 ( unsigned int *d_state_final_multi, unsigned char *d_text, unsigned int *d_out, size_t final_multi_pitch, int m, int n, int p_size, int alphabet, int numBlocks ) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int effective_final_multi_pitch = final_multi_pitch / sizeof ( int );
	
	int charactersPerBlock = n / numBlocks;
	
	int startBlock = blockIdx.x * charactersPerBlock;
	int stopBlock = startBlock + charactersPerBlock;
	
	int charactersPerThread = ( stopBlock - startBlock ) / blockDim.x;
	
	int startThread = startBlock + charactersPerThread * threadIdx.x + m - 1;
	int stopThread = startThread + charactersPerThread;

	int r, s;
	
	int column = startThread, i, j, l;
	
	while ( column < stopThread ) {
	
		r = 0;
		j = 0;

		while ( j < m && ( s = tex2D ( tex_state_transition, d_text[column - j], r ) ) != -1 ) {
				
			r = s;
			j++;
		}

		if ( j == m && tex2D ( tex_state_final_multi, 0, r ) > 0 ) {

				for ( i = 1; i <= tex2D ( tex_state_final_multi, 0, r ); i++ ) {

					//memcmp implementation
					for ( l = 0; l < m; l++ )
						if ( tex2D ( tex_pattern, l, d_state_final_multi[r * effective_final_multi_pitch + i] ) != d_text[column - m + 1 + l] )
							break;

					if ( l == m ) {
						d_out[idx]++;
						break;
					}

				}
					column++;
			}
			else	
				column += MAX ( m - j, 1);
	}
}
	
extern "C" void cuda_sbom2 ( unsigned char *pattern, int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_final_multi ) {

	//Pointer for device memory
	int *d_state_transition;
	unsigned int *d_state_final_multi, *d_out;
	
	unsigned char *d_pattern, *d_text;

	size_t pattern_pitch, transition_pitch, final_multi_pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m ) {
		printf("The text size is too small\n");
		exit(1);
	}

	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );

	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_pattern, &pattern_pitch, m * sizeof ( unsigned char ), p_size ) );
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &transition_pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	checkCudaErrors ( hipMallocPitch ( &d_state_final_multi, &final_multi_pitch, 200 * sizeof ( unsigned int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );

	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_pattern, pattern_pitch, pattern, m * sizeof ( unsigned char ), m * sizeof ( unsigned char ), p_size, hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, transition_pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), m * p_size + 1, hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy2D ( d_state_final_multi, final_multi_pitch, state_final_multi, 200 * sizeof ( int ), 200 * sizeof ( int ), m * p_size + 1, hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc int_desc = hipCreateChannelDesc<int>();
	hipChannelFormatDesc char_desc = hipCreateChannelDesc<unsigned char>();
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_transition, d_state_transition, int_desc, alphabet, m * p_size + 1, transition_pitch ) );
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_final_multi, d_state_final_multi, int_desc, 200, m * p_size + 1, final_multi_pitch ) );
	checkCudaErrors ( hipBindTexture2D ( 0, tex_pattern, d_pattern, char_desc, m, p_size, pattern_pitch ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;
	
	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//cudaPrintfInit();
	
	//Executing kernel in the device
	sbom_kernel2<<<dimGrid, dimBlock>>>( d_state_final_multi, d_text, d_out, final_multi_pitch, m, n, p_size, alphabet, numBlocks );
	
	checkCUDAError("kernel invocation");
	
	//cudaPrintfDisplay(stdout, true);
	//cudaPrintfEnd();

	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	hipEventDestroy ( start );
	hipEventDestroy ( stop );
	
	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	
  	//Look at the results
  	int i, matches = 0;
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf ("Kernel 2 matches \t%i\t time \t%f\n", matches, time/1000);
		
	//Free host and device memory
	free ( h_out );
	
	hipFree ( d_text );
	hipFree ( d_pattern );
	hipFree ( d_state_transition );
	hipFree ( d_state_final_multi );
	hipFree ( d_out );
}

__global__ void sbom_kernel1 ( int *d_state_transition, unsigned int *d_state_final_multi, unsigned char *d_pattern, unsigned char *d_text, unsigned int *d_out, size_t pattern_pitch, size_t transition_pitch, size_t final_multi_pitch, int m, int n, int p_size, int alphabet, int numBlocks ) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int effective_transition_pitch = transition_pitch / sizeof ( int );
	int effective_final_multi_pitch = final_multi_pitch / sizeof ( int );
	
	int charactersPerBlock = n / numBlocks;
	
	int startBlock = blockIdx.x * charactersPerBlock;
	int stopBlock = startBlock + charactersPerBlock;
	
	int charactersPerThread = ( stopBlock - startBlock ) / blockDim.x;
	
	int startThread = startBlock + charactersPerThread * threadIdx.x + m - 1;
	int stopThread = startThread + charactersPerThread;

	int r, s;
	
	int column = startThread, i, j, l;
	
	while ( column < stopThread ) {
	
		r = 0;
		j = 0;

		while ( j < m && ( s = d_state_transition[r * effective_transition_pitch + d_text[column - j]] ) != -1 ) {
				
			r = s;
			j++;
		}
		
		if ( j == m && d_state_final_multi[r * effective_final_multi_pitch] > 0 ) {

			for ( i = 1; i <= d_state_final_multi[r * effective_final_multi_pitch]; i++ ) {

				//memcmp implementation
				for ( l = 0; l < m; l++ )
					if ( d_pattern[d_state_final_multi[r * effective_final_multi_pitch + i] * pattern_pitch + l] != d_text[column - m + 1 + l] )
						break;

				if ( l == m ) {
					d_out[idx]++;
					break;
				}
			}
				column++;
		}
		else	
			column += MAX ( m - j, 1);
	}
}
	
extern "C" void cuda_sbom1 ( unsigned char *pattern, int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_final_multi ) {

	//Pointer for device memory
	int *d_state_transition;
	unsigned int *d_state_final_multi, *d_out;
	
	unsigned char *d_pattern, *d_text;

	size_t pattern_pitch, transition_pitch, final_multi_pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m ) {
		printf("The text size is too small\n");
		exit(1);
	}

	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );

	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_pattern, &pattern_pitch, m * sizeof ( unsigned char ), p_size ) );
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &transition_pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	checkCudaErrors ( hipMallocPitch ( &d_state_final_multi, &final_multi_pitch, 200 * sizeof ( unsigned int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );

	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_pattern, pattern_pitch, pattern, m * sizeof ( unsigned char ), m * sizeof ( unsigned char ), p_size, hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, transition_pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy2D ( d_state_final_multi, final_multi_pitch, state_final_multi, 200 * sizeof ( int ), 200 * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;
	
	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//cudaPrintfInit();
	
	//Executing kernel in the device
	sbom_kernel1<<<dimGrid, dimBlock>>>( d_state_transition, d_state_final_multi, d_pattern, d_text, d_out, pattern_pitch, transition_pitch, final_multi_pitch, m, n, p_size, alphabet, numBlocks );
	
	checkCUDAError("kernel invocation");
	
	//cudaPrintfDisplay(stdout, true);
	//cudaPrintfEnd();

	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	hipEventDestroy ( start );
	hipEventDestroy ( stop );
	
	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	
  	//Look at the results
  	int i, matches = 0;
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf ("Kernel 1 matches \t%i\t time \t%f\n", matches, time/1000);
		
	//Free host and device memory
	free ( h_out );
	
	hipFree ( d_text );
	hipFree ( d_pattern );
	hipFree ( d_state_transition );
	hipFree ( d_state_final_multi );
	hipFree ( d_out );
}

