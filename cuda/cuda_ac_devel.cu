#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "../smatcher.h"

void checkCUDAError(const char *msg) {
	
	hipError_t err = hipGetLastError();
	
	if( hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
		exit(EXIT_FAILURE);
	}
}

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors( hipError_t err, const char *file, const int line ) {
	
	if( hipSuccess != err) {
		fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n", file, line, (int)err, hipGetErrorString( err ) );
		exit(-1);
	}
}

texture<int, hipTextureType2D> tex_state_transition;
texture<unsigned int, hipTextureType1D> tex_state_supply;
texture<unsigned int, hipTextureType1D> tex_state_final;

//Optimization 6: Store the results to a temporary var and then move them to the global array with converged memory transactions
__global__ void ac_kernel7 ( int *d_state_transition, unsigned int *d_state_supply, unsigned int *d_state_final, unsigned char *d_text, unsigned int *d_out, int m, int n, int p_size, int alphabet, int numBlocks, int sharedMemSize ) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	int r, s;
	
	int i, j, k, column;
	
	int readsPerThread = sharedMemSize / ( blockDim.x * 16 );
	
	int startThread = readsPerThread * threadIdx.x;
	int stopThread = startThread + readsPerThread + ( m - 1 ) / 16 + 1;
	
	//Define space in shared memory
	//For every m - 1 multiple of 16, an additional uint4 should be reserved for redundancy
	extern __shared__ uint4 uint4_s_array[];

	//cast data to uint4
	uint4 *uint4_text = reinterpret_cast < uint4 * > ( d_text );
	uint4 uint4_var;
	
	//recast data to uchar4
	uchar4 c0, c4, c8, c12;
	unsigned char char_array[16];
	
	volatile register int matches = 0;
	
	//cuPrintf("start %i, stop %i\n", startThread, stopThread);
	
	for ( int globalMemIndex = blockIdx.x * sharedMemSize; globalMemIndex < n; globalMemIndex += numBlocks * sharedMemSize ) {
		
		for ( i = globalMemIndex / 16 + threadIdx.x, j = 0 + threadIdx.x; ( j < ( sharedMemSize + m - 1 ) / 16 + 1 && i < n / 16 ); i+=blockDim.x, j+=blockDim.x )
			uint4_s_array[j] = uint4_text[i];
			
		__syncthreads();
		
		r = 0;
		
		for ( column = startThread; column < stopThread && globalMemIndex + column * 16 < n; column++ ) {
			
			uint4_var = uint4_s_array[column];
			
			//recast data back to char after the memory transaction
			c0 = *reinterpret_cast<uchar4 *> ( &uint4_var.x );
			c4 = *reinterpret_cast<uchar4 *> ( &uint4_var.y );
			c8 = *reinterpret_cast<uchar4 *> ( &uint4_var.z );
			c12 = *reinterpret_cast<uchar4 *> ( &uint4_var.w );
			
			char_array[0] = c0.x;
			char_array[1] = c0.y;
			char_array[2] = c0.z;
			char_array[3] = c0.w;
			
			char_array[4] = c4.x;
			char_array[5] = c4.y;
			char_array[6] = c4.z;
			char_array[7] = c4.w;
			
			char_array[8] = c8.x;
			char_array[9] = c8.y;
			char_array[10] = c8.z;
			char_array[11] = c8.w;
			
			char_array[12] = c12.x;
			char_array[13] = c12.y;
			char_array[14] = c12.z;
			char_array[15] = c12.w;
			
			#pragma unroll 16
			for ( k = 0; ( k < 16 && column * 16 + k < stopThread * 16 + m - 1 ); k++ ) {
			
				while ( ( s = tex2D ( tex_state_transition, char_array[k], r ) ) == -1 )
					r = tex1Dfetch ( tex_state_supply, r );
				r = s;
			
				matches += tex1Dfetch ( tex_state_final, r );
			}
		}
		
		__syncthreads();
	}
	
	d_out[idx] = matches;
}


//Optimization 5: Store retrieved data from global memory to shared memory in a round-robin fashion to avoid bank conflicts
__global__ void ac_kernel6 ( int *d_state_transition, unsigned int *d_state_supply, unsigned int *d_state_final, unsigned char *d_text, unsigned int *d_out, int m, int n, int p_size, int alphabet, int numBlocks, int sharedMemSize ) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	int r, s;
	
	int i, j, k, column;
	
	int readsPerThread = sharedMemSize / ( blockDim.x * 4 );
	
	int startThread = readsPerThread * threadIdx.x;
	int stopThread = startThread + readsPerThread + ( m - 1 ) / 4 + 1;

	//Define space in shared memory
	//For every m - 1 multiple of 16, an additional uint4 should be reserved for redundancy
	extern __shared__ uchar4 uchar4_s_array[];
	
	//cast data to uint4
	uint4 *uint4_text = reinterpret_cast < uint4 * > ( d_text );
	uint4 uint4_var;
	
	//recast data to uchar4
	uchar4 c0, c4, c8, c12;
	unsigned char char_array[4];
	
	for ( int globalMemIndex = blockIdx.x * sharedMemSize; globalMemIndex < n; globalMemIndex += numBlocks * sharedMemSize ) {
		
		for ( i = globalMemIndex / 16 + threadIdx.x, j = 0 + threadIdx.x; ( j < ( sharedMemSize + m - 1 ) / 16 + 1 && i < n / 16 ); i+=blockDim.x, j+=blockDim.x ) {
		
			uint4_var = uint4_text[i];
		
			//recast data back to char after the memory transaction
			c0 = *reinterpret_cast<uchar4 *> ( &uint4_var.x );
			c4 = *reinterpret_cast<uchar4 *> ( &uint4_var.y );
			c8 = *reinterpret_cast<uchar4 *> ( &uint4_var.z );
			c12 = *reinterpret_cast<uchar4 *> ( &uint4_var.w );
		
			//Every 4 threads can write the 16 bytes beginning from same offset of the shared memory since they lie in different banks (4 * 16 = 64 bytes)
			//The next 4 threads should begin writing to an offset += 4 from the previous to start to a different bank
			
			//i % n =  i & ( n -1))
			int tid16 = threadIdx.x % 16;
			
			if ( tid16 < 4 ) {

				uchar4_s_array[threadIdx.x * 4 + 0] = c0;
				uchar4_s_array[threadIdx.x * 4 + 1] = c4;
				uchar4_s_array[threadIdx.x * 4 + 2] = c8;
				uchar4_s_array[threadIdx.x * 4 + 3] = c12;
				
			} else if ( tid16 < 8 ) {
			
				uchar4_s_array[threadIdx.x * 4 + 1] = c4;
				uchar4_s_array[threadIdx.x * 4 + 2] = c8;
				uchar4_s_array[threadIdx.x * 4 + 3] = c12;
				uchar4_s_array[threadIdx.x * 4 + 0] = c0;

			} else if ( tid16 < 12 ) {
			
				uchar4_s_array[threadIdx.x * 4 + 2] = c8;
				uchar4_s_array[threadIdx.x * 4 + 3] = c12;
				uchar4_s_array[threadIdx.x * 4 + 0] = c0;
				uchar4_s_array[threadIdx.x * 4 + 1] = c4;
				
			} else {
			
				uchar4_s_array[threadIdx.x * 4 + 3] = c12;
				uchar4_s_array[threadIdx.x * 4 + 0] = c0;
				uchar4_s_array[threadIdx.x * 4 + 1] = c4;
				uchar4_s_array[threadIdx.x * 4 + 2] = c8;
			}
		}
			
		//Add m - 1 redundant characters at the end of the shared memory
		//FIXME: optimize this!!
		//if ( threadIdx.x < m - 1 )
		//	s_array[sharedMemSize + threadIdx.x] = d_text[globalMemIndex + sharedMemSize + threadIdx.x]; 
			
		__syncthreads();
		
		r = 0;
		
		for ( column = startThread; ( column < stopThread && globalMemIndex + column * 4 < n ); column++ ) {
		
			c0 = uchar4_s_array[column];
		
			char_array[0] = c0.x;
			char_array[1] = c0.y;
			char_array[2] = c0.z;
			char_array[3] = c0.w;
			
			#pragma unroll 4
			for ( k = 0; ( k < 4 && column * 4 + k < stopThread * 4 + m - 1 ); k++ ) {
		
				while ( ( s = tex2D ( tex_state_transition, char_array[k], r ) ) == -1 )
					r = tex1Dfetch ( tex_state_supply, r );
				r = s;
				
				d_out[idx] += tex1Dfetch ( tex_state_final, r );
			}
		}
		
		__syncthreads();
	}
}

//Optimization 4: Store uint4s to shared memory and then extract them after reading to slightly increase shared memory throughput
__global__ void ac_kernel5 ( int *d_state_transition, unsigned int *d_state_supply, unsigned int *d_state_final, unsigned char *d_text, unsigned int *d_out, int m, int n, int p_size, int alphabet, int numBlocks, int sharedMemSize ) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	int r, s;
	
	int i, j, k, column;
	
	int readsPerThread = sharedMemSize / ( blockDim.x * 16 );
	
	int startThread = readsPerThread * threadIdx.x;
	int stopThread = startThread + readsPerThread + ( m - 1 ) / 16 + 1;
	
	//Define space in shared memory
	//For every m - 1 multiple of 16, an additional uint4 should be reserved for redundancy
	extern __shared__ uint4 uint4_s_array[];

	//cast data to uint4
	uint4 *uint4_text = reinterpret_cast < uint4 * > ( d_text );
	uint4 uint4_var;
	
	//recast data to uchar4
	uchar4 c0, c4, c8, c12;
	unsigned char char_array[16];
	
	//cuPrintf("start %i, stop %i\n", startThread, stopThread);
	
	for ( int globalMemIndex = blockIdx.x * sharedMemSize; globalMemIndex < n; globalMemIndex += numBlocks * sharedMemSize ) {
		
		for ( i = globalMemIndex / 16 + threadIdx.x, j = 0 + threadIdx.x; ( j < ( sharedMemSize + m - 1 ) / 16 + 1 && i < n / 16 ); i+=blockDim.x, j+=blockDim.x )
			uint4_s_array[j] = uint4_text[i];
			
		__syncthreads();
		
		r = 0;
		
		for ( column = startThread; column < stopThread && globalMemIndex + column * 16 < n; column++ ) {
			
			uint4_var = uint4_s_array[column];
			
			//recast data back to char after the memory transaction
			c0 = *reinterpret_cast<uchar4 *> ( &uint4_var.x );
			c4 = *reinterpret_cast<uchar4 *> ( &uint4_var.y );
			c8 = *reinterpret_cast<uchar4 *> ( &uint4_var.z );
			c12 = *reinterpret_cast<uchar4 *> ( &uint4_var.w );
			
			char_array[0] = c0.x;
			char_array[1] = c0.y;
			char_array[2] = c0.z;
			char_array[3] = c0.w;
			
			char_array[4] = c4.x;
			char_array[5] = c4.y;
			char_array[6] = c4.z;
			char_array[7] = c4.w;
			
			char_array[8] = c8.x;
			char_array[9] = c8.y;
			char_array[10] = c8.z;
			char_array[11] = c8.w;
			
			char_array[12] = c12.x;
			char_array[13] = c12.y;
			char_array[14] = c12.z;
			char_array[15] = c12.w;
			
			#pragma unroll 16
			for ( k = 0; ( k < 16 && column * 16 + k < stopThread * 16 + m - 1 ); k++ ) {
			
				while ( ( s = tex2D ( tex_state_transition, char_array[k], r ) ) == -1 )
					r = tex1Dfetch ( tex_state_supply, r );
				r = s;
			
				d_out[idx] += tex1Dfetch ( tex_state_final, r );
			}
		}
		
		__syncthreads();
	}
}

//Optimization 3: Read 16 byte words as uint4 from global to shared memory. This increases bandwidth utilization to 100%
__global__ void ac_kernel4 ( int *d_state_transition, unsigned int *d_state_supply, unsigned int *d_state_final, unsigned char *d_text, unsigned int *d_out, int m, int n, int p_size, int alphabet, int numBlocks, int sharedMemSize ) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	int r, s;
	
	int i, j, column;
	
	int charactersPerThread = sharedMemSize / blockDim.x;
	
	int startThread = charactersPerThread * threadIdx.x;
	int stopThread = startThread + charactersPerThread + m - 1;

	//Define space in shared memory
	extern __shared__ unsigned char s_array[];
	
	//cast data to uint4
	uint4 *uint4_text = reinterpret_cast < uint4 * > ( d_text );
	uint4 uint4_var;
	
	//recast data to uchar4
	uchar4 c0, c4, c8, c12;
	
	for ( int globalMemIndex = blockIdx.x * sharedMemSize; globalMemIndex < n; globalMemIndex += numBlocks * sharedMemSize ) {
		
		for ( i = globalMemIndex/16 + threadIdx.x, j = 0 + threadIdx.x; j < sharedMemSize / 16 && i < n / 16; i+=blockDim.x, j+=blockDim.x ) {
			
			uint4_var = uint4_text[i];
			
			//recast data back to char after the memory transaction
			c0 = *reinterpret_cast<uchar4 *> ( &uint4_var.x );
			c4 = *reinterpret_cast<uchar4 *> ( &uint4_var.y );
			c8 = *reinterpret_cast<uchar4 *> ( &uint4_var.z );
			c12 = *reinterpret_cast<uchar4 *> ( &uint4_var.w );

			s_array[j * 16 + 0] = c0.x;
                        s_array[j * 16 + 1] = c0.y;
                        s_array[j * 16 + 2] = c0.z;
                        s_array[j * 16 + 3] = c0.w;
                        
                        s_array[j * 16 + 4] = c4.x;
                        s_array[j * 16 + 5] = c4.y;
                        s_array[j * 16 + 6] = c4.z;
                        s_array[j * 16 + 7] = c4.w;
                        
                        s_array[j * 16 + 8] = c8.x;
                        s_array[j * 16 + 9] = c8.y;
                        s_array[j * 16 + 10] = c8.z;
                        s_array[j * 16 + 11] = c8.w;
                        
                        s_array[j * 16 + 12] = c12.x;
                        s_array[j * 16 + 13] = c12.y;
                        s_array[j * 16 + 14] = c12.z;
                        s_array[j * 16 + 15] = c12.w;
		}

		//Add m - 1 redundant characters at the end of the shared memory
		//FIXME: optimize this!!
		if ( threadIdx.x < m - 1 )
			s_array[sharedMemSize + threadIdx.x] = d_text[globalMemIndex + sharedMemSize + threadIdx.x]; 
			
		__syncthreads();
		
		r = 0;
		
		for ( column = startThread; ( column < stopThread && globalMemIndex + column < n ); column++ ) {
		
			while ( ( s = tex2D ( tex_state_transition, s_array[column], r ) ) == -1 )
				r = tex1Dfetch ( tex_state_supply, r );
			r = s;
			
			d_out[idx] += tex1Dfetch ( tex_state_final, r );
		}
		
		__syncthreads();
	}
}

//Optimization 2: Read sharedMemSize characters byte-to-byte from global memory to shared memory to coalescelce memory transactions 
__global__ void ac_kernel3 ( int *d_state_transition, unsigned int *d_state_supply, unsigned int *d_state_final, unsigned char *d_text, unsigned int *d_out, int m, int n, int p_size, int alphabet, int numBlocks, int sharedMemSize ) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	int r, s;
	
	int i, j, column;
	
	int charactersPerThread = sharedMemSize / blockDim.x;
	
	int startThread = charactersPerThread * threadIdx.x;
	int stopThread = startThread + charactersPerThread + m - 1;

	//Define space in shared memory
	extern __shared__ unsigned char s_array[];
	
	for ( int globalMemIndex = blockIdx.x * sharedMemSize; globalMemIndex < n; globalMemIndex += numBlocks * sharedMemSize ) {
	
		/*if ( threadIdx.x == 0 )
			for ( i = globalMemIndex, j = 0; ( j < sharedMemSize + m - 1 && i < n ); i++, j++ )
				s_array[j] = d_text[i];
		*/
		
		for ( i = globalMemIndex + threadIdx.x, j = 0 + threadIdx.x; ( j < sharedMemSize + m - 1 && i < n ); i+=blockDim.x, j+=blockDim.x )
			s_array[j] = d_text[i];
			
		__syncthreads();
		
		r = 0;
		
		for ( column = startThread; ( column < stopThread && globalMemIndex + column < n ); column++ ) {
		
			while ( ( s = tex2D ( tex_state_transition, s_array[column], r ) ) == -1 )
				r = tex1Dfetch ( tex_state_supply, r );
			r = s;
			
			d_out[idx] += tex1Dfetch ( tex_state_final, r );
		}
		
		__syncthreads();
	}
}

//Optimization 1: Use the texture cache for the pattern
__global__ void ac_kernel2 ( int *d_state_transition, unsigned int *d_state_supply, unsigned int *d_state_final, unsigned char *d_text, unsigned int *d_out, int m, int n, int p_size, int alphabet, int numBlocks ) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	int charactersPerBlock = n / numBlocks;
	
	int startBlock = blockIdx.x * charactersPerBlock;
	int stopBlock = startBlock + charactersPerBlock;
	
	int charactersPerThread = ( stopBlock - startBlock ) / blockDim.x;
	
	int startThread = startBlock + charactersPerThread * threadIdx.x;
	int stopThread = startThread + charactersPerThread + m - 1;
	
	int r = 0, s;
	
	int column;
	
	for ( column = startThread; ( column < stopThread && column < n ); column++ ) {

		while ( ( s = tex2D ( tex_state_transition, d_text[column], r ) ) == -1 )
			r = tex1Dfetch ( tex_state_supply, r );
		r = s;
			
		d_out[idx] += tex1Dfetch ( tex_state_final, r );
	}
}

__global__ void ac_kernel1 ( int *d_state_transition, unsigned int *d_state_supply, unsigned int *d_state_final, unsigned char *d_text, unsigned int *d_out, size_t pitch, int m, int n, int p_size, int alphabet, int numBlocks ) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int effective_pitch = pitch / sizeof ( int );
	
	int charactersPerBlock = n / numBlocks;
	
	int startBlock = blockIdx.x * charactersPerBlock;
	int stopBlock = startBlock + charactersPerBlock;
	
	int charactersPerThread = ( stopBlock - startBlock ) / blockDim.x;
	
	int startThread = startBlock + charactersPerThread * threadIdx.x;
	int stopThread = startThread + charactersPerThread + m - 1;

	int r = 0, s;
	
	int column;
	
	//cuPrintf("Working from %i to %i chars %i\n", startThread, stopThread, charactersPerThread);
	
	for ( column = startThread; ( column < stopThread && column < n ); column++ ) {

		while ( ( s = d_state_transition[r * effective_pitch + d_text[column]] ) == -1 )
			r = d_state_supply[r];
		r = s;
			
		d_out[idx] += d_state_final[r];
	}
}

void fail ( const char * format ) {

	printf("Error: %s", format);
	exit ( 1 );
}

extern "C" void cuda_ac7 ( unsigned char **pattern, int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_supply, unsigned int *state_final ) {

	//Pointer for device memory
	int *d_state_transition;
	unsigned int *d_state_supply, *d_state_final, *d_out;
	
	unsigned char *d_text;

	size_t pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256, sharedMemSize = 16128;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m )
		fail("The text size is too small\n");
	
	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	
	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_supply, state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_final, state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	
	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_transition, d_state_transition, desc, alphabet * sizeof ( int ), ( m * p_size + 1 ), pitch ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_supply, d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_final, d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;

	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//Executing kernel in the device
	ac_kernel7<<<dimGrid, dimBlock, sharedMemSize + 16 * ( ( m - 1 ) / 16 + 1 )>>>( d_state_transition, d_state_supply, d_state_final, d_text, d_out, m, n, p_size, alphabet, numBlocks, sharedMemSize );
	checkCUDAError("kernel invocation");
	
	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	printf ("Time for kernel 7: %f sec\n", time/1000);

	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	   
  	//Look at the results
  	int i, matches = 0;
  	
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf("cuda matches: %i\n", matches);
	
	//Free host and device memory
	free ( h_out );

	hipFree ( d_text );
	hipFree ( d_state_transition );
	hipFree ( d_state_supply );
	hipFree ( d_state_final );
	hipFree ( d_out );
}

extern "C" void cuda_ac6 ( unsigned char **pattern, int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_supply, unsigned int *state_final ) {

	//Pointer for device memory
	int *d_state_transition;
	unsigned int *d_state_supply, *d_state_final, *d_out;
	
	unsigned char *d_text;

	size_t pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256, sharedMemSize = 16128;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m )
		fail("The text size is too small\n");
	
	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	
	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_supply, state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_final, state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	
	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_transition, d_state_transition, desc, alphabet * sizeof ( int ), ( m * p_size + 1 ), pitch ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_supply, d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_final, d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;

	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//Executing kernel in the device
	ac_kernel6<<<dimGrid, dimBlock, sharedMemSize + 16 * ( ( m - 1 ) / 16 + 1 )>>>( d_state_transition, d_state_supply, d_state_final, d_text, d_out, m, n, p_size, alphabet, numBlocks, sharedMemSize );
	checkCUDAError("kernel invocation");
	
	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	printf ("Time for kernel 6: %f sec\n", time/1000);

	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	   
  	//Look at the results
  	int i, matches = 0;
  	
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf("cuda matches: %i\n", matches);
	
	//Free host and device memory
	free ( h_out );

	hipFree ( d_text );
	hipFree ( d_state_transition );
	hipFree ( d_state_supply );
	hipFree ( d_state_final );
	hipFree ( d_out );
}

extern "C" void cuda_ac5 ( unsigned char **pattern, int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_supply, unsigned int *state_final ) {

	//Pointer for device memory
	int *d_state_transition;
	unsigned int *d_state_supply, *d_state_final, *d_out;
	
	unsigned char *d_text;

	size_t pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256, sharedMemSize = 16128;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m )
		fail("The text size is too small\n");
	
	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	
	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_supply, state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_final, state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	
	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_transition, d_state_transition, desc, alphabet * sizeof ( int ), ( m * p_size + 1 ), pitch ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_supply, d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_final, d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;

	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//Executing kernel in the device
	ac_kernel5<<<dimGrid, dimBlock, sharedMemSize + 16 * ( ( m - 1 ) / 16 + 1 )>>>( d_state_transition, d_state_supply, d_state_final, d_text, d_out, m, n, p_size, alphabet, numBlocks, sharedMemSize );
	checkCUDAError("kernel invocation");
	
	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	printf ("Time for kernel 5: %f sec\n", time/1000);

	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	   
  	//Look at the results
  	int i, matches = 0;
  	
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf("cuda matches: %i\n", matches);
	
	//Free host and device memory
	free ( h_out );

	hipFree ( d_text );
	hipFree ( d_state_transition );
	hipFree ( d_state_supply );
	hipFree ( d_state_final );
	hipFree ( d_out );
}

extern "C" void cuda_ac4 ( unsigned char **pattern, int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_supply, unsigned int *state_final ) {

	//Pointer for device memory
	int *d_state_transition;
	unsigned int *d_state_supply, *d_state_final, *d_out;
	
	unsigned char *d_text;

	size_t pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256, sharedMemSize = 16128;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m )
		fail("The text size is too small\n");
	
	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	
	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_supply, state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_final, state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	
	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_transition, d_state_transition, desc, alphabet * sizeof ( int ), ( m * p_size + 1 ), pitch ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_supply, d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_final, d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;

	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//Executing kernel in the device
	ac_kernel4<<<dimGrid, dimBlock, sharedMemSize + m - 1>>>( d_state_transition, d_state_supply, d_state_final, d_text, d_out, m, n, p_size, alphabet, numBlocks, sharedMemSize );
	checkCUDAError("kernel invocation");
	
	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	printf ("Time for kernel 4: %f sec\n", time/1000);

	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	   
  	//Look at the results
  	int i, matches = 0;
  	
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf("cuda matches: %i\n", matches);
	
	//Free host and device memory
	free ( h_out );

	hipFree ( d_text );
	hipFree ( d_state_transition );
	hipFree ( d_state_supply );
	hipFree ( d_state_final );
	hipFree ( d_out );
}

extern "C" void cuda_ac3 ( unsigned char **pattern, int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_supply, unsigned int *state_final ) {

	//Pointer for device memory
	int *d_state_transition;
	unsigned int *d_state_supply, *d_state_final, *d_out;
	
	unsigned char *d_text;

	size_t pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256, sharedMemSize = 16128;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m )
		fail("The text size is too small\n");
	
	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	
	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_supply, state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_final, state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	
	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_transition, d_state_transition, desc, alphabet * sizeof ( int ), ( m * p_size + 1 ), pitch ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_supply, d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_final, d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;

	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//Executing kernel in the device
	ac_kernel3<<<dimGrid, dimBlock, sharedMemSize + m - 1>>>( d_state_transition, d_state_supply, d_state_final, d_text, d_out, m, n, p_size, alphabet, numBlocks, sharedMemSize );
	checkCUDAError("kernel invocation");
	
	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	printf ("Time for kernel 3: %f sec\n", time/1000);

	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	   
  	//Look at the results
  	int i, matches = 0;
  	
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf("cuda matches: %i\n", matches);
	
	//Free host and device memory
	free ( h_out );

	hipFree ( d_text );
	hipFree ( d_state_transition );
	hipFree ( d_state_supply );
	hipFree ( d_state_final );
	hipFree ( d_out );
}

extern "C" void cuda_ac2 ( unsigned char **pattern, int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_supply, unsigned int *state_final ) {

	//Pointer for device memory
	int *d_state_transition;
	unsigned int *d_state_supply, *d_state_final, *d_out;
	
	unsigned char *d_text;

	size_t pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m )
		fail("The text size is too small\n");
	
	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );

	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_supply, state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_final, state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );

	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_transition, d_state_transition, desc, alphabet * sizeof ( int ), ( m * p_size + 1 ), pitch ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_supply, d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_final, d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;

	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//Executing kernel in the device
	ac_kernel2<<<dimGrid, dimBlock>>>( d_state_transition, d_state_supply, d_state_final, d_text, d_out, m, n, p_size, alphabet, numBlocks );
	checkCUDAError("kernel invocation");
	
	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	printf ("Time for kernel 2: %f sec\n", time/1000);

	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
   
  	//Look at the results
  	int i, matches = 0;
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf("cuda matches: %i\n", matches);
		
	//Free host and device memory
	free ( h_out );
	
	hipFree ( d_text );
	hipFree ( d_state_transition );
	hipFree ( d_state_supply );
	hipFree ( d_state_final );
	hipFree ( d_out );
}

extern "C" void cuda_ac1 ( unsigned char **pattern, int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_supply, unsigned int *state_final ) {

	//Pointer for device memory
	int *d_state_transition;
	unsigned int *d_state_supply, *d_state_final, *d_out;
	
	unsigned char *d_text;

	size_t pitch;
	
	int numBlocks = 30, numThreadsPerBlock = 256;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m )
		fail("The text size is too small\n");
	
	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );

	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_supply, state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_final, state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );

	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;
	
	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//cudaPrintfInit();
	
	//Executing kernel in the device
	ac_kernel1<<<dimGrid, dimBlock>>>( d_state_transition, d_state_supply, d_state_final, d_text, d_out, pitch, m, n, p_size, alphabet, numBlocks );
	checkCUDAError("kernel invocation");
	
	//cudaPrintfDisplay(stdout, true);
	//cudaPrintfEnd();

	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	printf ("Time for kernel 1: %f sec\n", time/1000);
	
	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	
  	//Look at the results
  	int i, matches = 0;
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf("cuda matches: %i\n", matches);
		
	//Free host and device memory
	free ( h_out );
	
	hipFree ( d_text );
	hipFree ( d_state_transition );
	hipFree ( d_state_supply );
	hipFree ( d_state_final );
	hipFree ( d_out );
}

